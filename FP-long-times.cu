#include "hip/hip_runtime.h"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>
#include "Random123/philox.h"
#include "Random123/u01.h"
#include "histo.h"



typedef r123::Philox2x32 RNG; // un counter-based RNG

// check times
 #ifdef OMP
 #include "cpu_timer.h"
 #else
 #include "gpu_timer.h"
 #endif


/*Random number (gaussian distribution) - generated using philox*/
__device__
float box_muller(RNG::ctr_type r_philox)
{
	// transforma el philox number a dos uniformes en (0,1]
 	float u1 = u01_open_closed_32_53(r_philox[0]);
  float u2 = u01_open_closed_32_53(r_philox[1]);

  float r = sqrtf( -2.0*logf(u1) );
  float theta = 2.0*M_PI*u2;
	
  return r*sinf(theta);    			
}


#define Dt		0.001  // paso temporal
#define NROBINS		100   // numero de bins
#define NROPARTS	1000000 // numero de particulas	
#define SEED		98273614 // una semilla global 

// force
__device__ float Fuerza(float x)
{
	// return -d/dx [sinf(2 M_PI x) + 0.25 sinf(4 M_PI x)]
	return -M_PI*(2.0f*cosf(2.0f*M_PI*x) + cosf(4.0f*M_PI*x));
}


// this functor to evolve the dynamic of the brownian particle
// trun steps at temperature T
struct dinamica
{ // time steps
  /* iteration - histogram */
  unsigned int trun; // iterations in histogram 
  /* Iteraciones total */
  unsigned int t;    // total iterations ~ time 

  float T;	      // temperature

  /* fac --> sqrt ( 2*T*delta t )*/
  float fac;         // aux variable
  dinamica(float _T, unsigned int _trun, unsigned int _t):T(_T),trun(_trun),t(_t)
  {
	fac=sqrtf(2.0f*T*Dt);
  };	

  // tid as counter to identify the particle, and position x
  // update x
  __device__
  float operator()(unsigned int tid, float x)
  {
    // keys and counters 
    RNG philox; 	
    RNG::ctr_type c={{}};
    RNG::key_type k={{}};
    RNG::ctr_type r;

    // tid que es el contador
    c[1]=SEED;
    c[0]=tid+SEED;
    k[0]=tid; 

    //generate random seq - SEED
    

    for(unsigned int i = 0; i < trun; ++i){
      c[0]=i+t;
      r = philox(c, k);
      
      //gaussian generator
      float randGauss=box_muller(r);
      
      //dynamics equation
      x = x + Fuerza(x)*Dt+ fac*randGauss;
      
      //periodic boundary conditions
      if(x<0.0f) x+=1.0;  // + 1
      if(x>1.0f) x-=1.0;  // - 1   --> x in range (0,1]
    }
    return x; 	
  }
};



#include <omp.h>

// to avoid "thrust::"
using namespace thrust;	
