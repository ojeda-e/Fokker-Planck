#include "hip/hip_runtime.h"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>
#include "Random123/philox.h"
#include "Random123/u01.h"
#include "histo.h"



typedef r123::Philox2x32 RNG; // un counter-based RNG

// check times
 #ifdef OMP
 #include "cpu_timer.h"
 #else
 #include "gpu_timer.h"
 #endif


/*Random number (gaussian distribution) - generated using philox*/
__device__
float box_muller(RNG::ctr_type r_philox)
{
	// transforma el philox number a dos uniformes en (0,1]
 	float u1 = u01_open_closed_32_53(r_philox[0]);
  float u2 = u01_open_closed_32_53(r_philox[1]);

  float r = sqrtf( -2.0*logf(u1) );
  float theta = 2.0*M_PI*u2;
	
  return r*sinf(theta);    			
}


#define Dt		0.001  // paso temporal
#define NROBINS		100   // numero de bins
#define NROPARTS	1000000 // numero de particulas	
#define SEED		98273614 // una semilla global 

// force
__device__ float Fuerza(float x)
{
	// return -d/dx [sinf(2 M_PI x) + 0.25 sinf(4 M_PI x)]
	return -M_PI*(2.0f*cosf(2.0f*M_PI*x) + cosf(4.0f*M_PI*x));
}


// this functor to evolve the dynamic of the brownian particle
// trun steps at temperature T
struct dinamica
{ // time steps
  /* iteration - histogram */
  unsigned int trun; // iterations in histogram 
  /* Iteraciones total */
  unsigned int t;    // total iterations ~ time 

  float T;	      // temperature

  /* fac --> sqrt ( 2*T*delta t )*/
  float fac;         // aux variable
  dinamica(float _T, unsigned int _trun, unsigned int _t):T(_T),trun(_trun),t(_t)
  {
	fac=sqrtf(2.0f*T*Dt);
  };	

  // tid as counter to identify the particle, and position x
  // update x
  __device__
  float operator()(unsigned int tid, float x)
  {
    // keys and counters 
    RNG philox; 	
    RNG::ctr_type c={{}};
    RNG::key_type k={{}};
    RNG::ctr_type r;

    // tid que es el contador
    c[1]=SEED;
    c[0]=tid+SEED;
    k[0]=tid; 

    //generate random seq - SEED
    

    for(unsigned int i = 0; i < trun; ++i){
      c[0]=i+t;
      r = philox(c, k);
      
      //gaussian generator
      float randGauss=box_muller(r);
      
      //dynamics equation
      x = x + Fuerza(x)*Dt+ fac*randGauss;
      
      //periodic boundary conditions
      if(x<0.0f) x+=1.0;  // + 1
      if(x>1.0f) x-=1.0;  // - 1   --> x in range (0,1]
    }
    return x; 	
  }
};



#include <omp.h>

// to avoid "thrust::"
using namespace thrust;	


int main(void)
{

  #ifdef OMP
  // print max num threads in version openMP
  std::cout << "#host OMP threads = " << omp_get_max_threads() << std::endl;
  cpu_timer timer;
  #else
  int card;
  hipGetDevice(&card);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, card);
  std::cout << "\nDevice Selected " << card << " " << deviceProp.name << "\n";
  gpu_timer timer; 
  #endif	

  // declare vector X (particle #) single precision
  device_vector<float> X(NROPARTS);

  // Initialize:

  // (1) Origin all:
  thrust::fill(X.begin(), X.end(), 0.);

  // (2) Uniform in range x = [0,1):
  //thrust::sequence(X.begin(), X.end());

  //(3) Random
  //thrust::generate(X.begin(), X.end(), rand);

  //declare Histogram in device, single precision
  device_vector<float> Histogram(NROBINS);

  float Temp=0.75f; // temperature
  unsigned trun=10; // step - histograms
  unsigned tiempo=0; // time (abs)

  // save to file
  std::ofstream histout("histograms.dat");

  // counter
  timer.tic();

  for(unsigned n=0;n<1000;n++,tiempo+=trun)
  {
    dense_histogram_data_on_device(X, Histogram, float(0.0), float(1.0));
	print_histograma(Histogram, float(0.0), float(1.0), histout);

    transform(
              counting_iterator<int>(0),
              counting_iterator<int>(NROPARTS),
              X.begin(),
              X.begin(),
              dinamica(Temp,trun,tiempo)
    );

  timer.tac();
  std::cout  << timer.ms_elapsed << "ms \n";

  }
  return 0;
}
